#include <hip/hip_runtime_api.h>
#include <benchmark/benchmark.h>
#include "cpptensor/tensor/tensor.hpp"
#include "cpptensor/backend/cuda_backend.hpp"
#include "cpptensor/dispatcher/kernelRegistry.h"

using namespace cpptensor;

static void BM_Add_CUDA(benchmark::State& state) {
    KernelRegistry::instance().registerKernel(OpType::Add, DeviceType::CUDA, CUDA::addKernel);
    Tensor A({2048, 2048}, 5.f, false, DeviceType::CUDA);
    Tensor B({2048, 2048}, 5.f, false, DeviceType::CUDA);
    for (auto _ : state) {
        Tensor C = A + B;
        benchmark::DoNotOptimize(C);
        hipDeviceSynchronize();
    }
}

static void BM_Mul_CUDA(benchmark::State& state) {
    KernelRegistry::instance().registerKernel(OpType::Mul, DeviceType::CUDA, CUDA::mulKernel);
    Tensor A({2048, 2048}, 5.f, false, DeviceType::CUDA);
    Tensor B({2048, 2048}, 5.f, false, DeviceType::CUDA);
    for (auto _ : state) {
        Tensor C = A * B;
        benchmark::DoNotOptimize(C);
        hipDeviceSynchronize();
    }
}

BENCHMARK(BM_Add_CUDA);
BENCHMARK(BM_Mul_CUDA);
BENCHMARK_MAIN();
